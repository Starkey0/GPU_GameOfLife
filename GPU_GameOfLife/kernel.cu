#include "hip/hip_runtime.h"
#include "kernel.h"

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

__global__ void evolveKernel(unsigned int cells[MAX_GRID_X*MAX_GRID_Y], unsigned int newcells[MAX_GRID_X*MAX_GRID_Y])
{
    unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int x,y;

    y = tid / MAX_GRID_X;
    x = tid - y* MAX_GRID_X;
    //printf("%d, %d, %d \n", tid, x, y);
    //newcells[x + y * MAX_GRID_X] = cells[x + y * MAX_GRID_X];


    if (x >= MAX_GRID_X || y >= MAX_GRID_Y)return;
    int n = 0;
    for (unsigned int y1 = y - 1; y1 <= y + 1; y1++)
        for (unsigned int x1 = x - 1; x1 <= x + 1; x1++)
            //if (!cells[(x1 + MAX_GRID_X) % MAX_GRID_X][(y1 + MAX_GRID_Y) % MAX_GRID_Y])
                //n++;
            if (!cells[(((x1 + MAX_GRID_X) % MAX_GRID_X) + ((y1 + MAX_GRID_Y) % MAX_GRID_Y) * MAX_GRID_X)])
                n++;

    if (!cells[x + y * MAX_GRID_X]) n--;
    newcells[x + y * MAX_GRID_X] = (n == 3 || (n == 2 && !cells[x + y * MAX_GRID_X])) > 0 ? 0 : 255;

    
}


// Helper function for using CUDA to add vectors in parallel.
extern "C" hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}

extern "C" hipError_t evolveWithCuda(unsigned int h_cells[MAX_GRID_X*MAX_GRID_Y])
{
    hipError_t cudaStatus;
    unsigned int *dA, *dB;
    size_t pitch;
    
    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    //hipMallocPitch(&dA, &pitch, sizeof(unsigned int) * MAX_GRID_X, MAX_GRID_Y);
    hipMalloc(&dA, sizeof(unsigned int) * MAX_GRID_X * MAX_GRID_Y);
    hipMalloc(&dB, sizeof(unsigned int) * MAX_GRID_X * MAX_GRID_Y);

    hipMemcpy(dA, h_cells, sizeof(unsigned int) * MAX_GRID_X * MAX_GRID_Y, hipMemcpyHostToDevice);

    int threadsperblock = TPB;
    int blockspergrid = MAX_GRID_X * MAX_GRID_Y / threadsperblock;

    // Launch a kernel on the GPU with one thread for each element.
    evolveKernel<<<blockspergrid, threadsperblock >>>(dA, dB);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "evolveKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    hipMemcpy(h_cells, dB, sizeof(unsigned int) * MAX_GRID_X * MAX_GRID_Y, hipMemcpyDeviceToHost);

    hipFree(dA);
    hipFree(dB);

Error:

    return cudaStatus;
}
